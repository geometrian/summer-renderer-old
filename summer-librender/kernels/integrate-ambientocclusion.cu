#include "hip/hip_runtime.h"
#pragma once


#include "generic-forward.cu"


namespace Summer {


class TraceInfoAmbientOcclusion final {
	public:
		RNG*const rng;

		float visibility;

	public:
		__device__ explicit TraceInfoAmbientOcclusion(RNG* rng) : rng(rng) {}
};


extern "C" __global__ void __raygen__ambientocclusion() {
	TraceInfoBasic trace_info;
	generic_forward0_raygen<TraceInfoBasic>(trace_info);
}


extern "C" __global__ void __miss__ambientocclusion_normal() {
	TraceInfoBasic const* trace_info = generic_forward0_miss<TraceInfoBasic>();

	semiAtomicAdd(
		interface.camera.framebuffer.layers.lighting_integration + trace_info->index.pixel_flat,
		Vec4f( Vec3f(1.0f), 1.0f )
	);
}
extern "C" __global__ void __miss__ambientocclusion_shadow() {
	TraceInfoAmbientOcclusion* trace_info = PackedPointer<TraceInfoAmbientOcclusion>::from_payloads01();

	trace_info->visibility = 1.0f;
}


extern "C" __global__ void __anyhit__ambientocclusion_normal() {
	TraceInfoBasic const* trace_info = PackedPointer<TraceInfoBasic>::from_payloads01();

	ShadingOperation shade_op(trace_info->rng);

	generic_forward0_anyhit<TraceInfoBasic>(shade_op,trace_info);
}
extern "C" __global__ void __anyhit__ambientocclusion_shadow() {
	TraceInfoAmbientOcclusion* trace_info = PackedPointer<TraceInfoAmbientOcclusion>::from_payloads01();

	ShadingOperation shade_op(trace_info->rng);

	if (shade_op.stochastic_is_opaque()) {
		trace_info->visibility = 0.0f;
		optixTerminateRay();
	} else {
		optixIgnoreIntersection();
	}
}


extern "C" __global__ void __closesthit__ambientocclusion_normal() {
	TraceInfoBasic* trace_info = PackedPointer<TraceInfoBasic>::from_payloads01();

	ShadingOperation shade_op(trace_info->rng);
	shade_op.compute_shade_info_pos_normals();

	generic_forward0_closesthit(shade_op,trace_info);

	Ray ray_shad = {
		shade_op.shade_info.pos_wld,
		trace_info->rng->get_coshemi(shade_op.shade_info.Nshad_wld)
	};
	offset_ray_orig( &ray_shad, shade_op.shade_info.Ngeom_wld );

	TraceInfoAmbientOcclusion trace_info_shad(trace_info->rng);
	PackedPointer<TraceInfoAmbientOcclusion> ptr = &trace_info_shad;
	optixTrace(
		interface.traversable,

		to_float3(ray_shad.orig), to_float3(ray_shad.dir),

		0.0f, 4.0f,//std::numeric_limits<float>::infinity(),
		0.0f,

		OptixVisibilityMask(0b11111111),

		OptixRayFlags::OPTIX_RAY_FLAG_NONE,
		1u, unsigned int(SUMMER_MAX_RAYTYPES),
		1u,

		ptr[0], ptr[1]
	);

	Vec4f color = Vec4f(Vec3f(trace_info_shad.visibility),1.0f);
	semiAtomicAdd(interface.camera.framebuffer.layers.lighting_integration+trace_info->index.pixel_flat,color);
}

  
}
